#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipblas.h>
#include <assert.h>
using namespace std;

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
    exit(-1);}} while(0)
#define CUDA_CALL(X) ERR_NE((X),hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

template<class T>
struct reCuBuffer
{
    T* data = NULL;
    int len = 0;
};

template<class T>
void resize(reCuBuffer<T>& buffer, int size)
{
    if(size > buffer.len)
    {
        if(buffer.len > 0)
            CUDA_CALL(hipFree(buffer.data));
            
        CUDA_CALL(hipMalloc( &(buffer.data), size));
        buffer.len = size;
    }
}

#define num_device 16

static reCuBuffer<int>   nnzPerCol_[num_device], ColInd_[num_device], RowPtr_[num_device];
static reCuBuffer<float> csrVal_[num_device], tranBuffer_[num_device];

struct cublasHandle_
{
    hipblasHandle_t handle_;
    bool init = false;
};
static cublasHandle_ handle2_[num_device];

void sparse_mm_dense_cusparse_backend(const int & cuda_device_id, const int & m, const int & n, const int & p, float * dA, float * dB, float * dC)
{
    assert(cuda_device_id>=0);
    hipSetDevice(cuda_device_id);
    reCuBuffer<int>& nnzPerCol    = nnzPerCol_[cuda_device_id];
    reCuBuffer<int>& ColInd       = ColInd_[cuda_device_id];
    reCuBuffer<int>& RowPtr       = RowPtr_[cuda_device_id];
    reCuBuffer<float>& csrVal     = csrVal_[cuda_device_id];
    reCuBuffer<float>& tranBuffer = tranBuffer_[cuda_device_id];

    // CT = A * BT
    resize(tranBuffer, m * p * sizeof(float));

    hipsparseHandle_t  handle;
    CUSPARSE_CALL(hipsparseCreate(&handle));

    // transform dense A to csr
    hipsparseMatDescr_t descrX;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrX));

    int total_nnz;
    resize(nnzPerCol, m * sizeof(int));

    CUSPARSE_CALL(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, n, m, descrX, dA, n, nnzPerCol.data, &total_nnz));
    resize(csrVal, total_nnz * sizeof(float));
    resize(ColInd, total_nnz * sizeof(int));
    resize(RowPtr, (m+1) * sizeof(int));

    CUSPARSE_CALL(hipsparseSdense2csc(handle, n, m, descrX, dA, n, nnzPerCol.data, csrVal.data, ColInd.data, RowPtr.data));

    // B * C
    hipsparseMatDescr_t descrA;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CALL(hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO));

    float alpha = 1.0f;
    float beta  = 0.0f;
    CUSPARSE_CALL(hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_TRANSPOSE,
        m,p,n,total_nnz,&alpha,descrA,csrVal.data,RowPtr.data, ColInd.data,dB,p,&beta,tranBuffer.data,m));

    // hipblasDestroy will synchronize the device
    hipblasHandle_t& handle2 = handle2_[cuda_device_id].handle_;
    if(!handle2_[cuda_device_id].init)
    {
        hipblasCreate(&handle2);
        handle2_[cuda_device_id].init = true;
    }

    // C need TRANSPOSE
    hipblasSgeam(handle2, HIPBLAS_OP_T, HIPBLAS_OP_T, p, m, &alpha, tranBuffer.data, m, &beta, tranBuffer.data, m, dC, p);
    //hipblasDestroy(handle2);

    CUSPARSE_CALL(hipsparseDestroy(handle));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrX));
    CUSPARSE_CALL(hipsparseDestroyMatDescr(descrA));
}
